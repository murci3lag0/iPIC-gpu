#include "include/numerics_gpu.h"

void set_current_gpu(int nb_fields, int tpb_fields, int ncell, float *Jx, float *Jy, float *Jz){
    set_current<<<nb_fields,tpb_fields>>>(ncell, Jx, Jy, Jz);
}

void move_part_gpu(int nb_part, int tpb_part, int npart, float dt, float Lx, float *x, float *u){
    move_part<<<nb_part,tpb_part>>>(npart, dt, Lx, x, u);
}

void add_current_gpu(int nb_part, int tpb_part, int npart, float wpi, float qm, float Lx, float dx, float *Jx, float *x, float *u){
    add_current<<<nb_part,tpb_part>>>(npart, wpi, qm, Lx, dx, Jx, x, u);
}

void calc_field_gpu(int nb_fields, int tpb_fields, int ncell, float dt, float *Jx, float *Ex){
    calc_field<<<nb_fields,tpb_fields>>>(ncell, dt, Jx, Ex);
}

void fields2part_gpu(int nb_part, int tpb_part, int npart, float dx, float *xp, float *Epx, float *Ecx){
    fields2part<<<nb_part,tpb_part>>>(npart, dx, xp, Epx, Ecx);
}

void update_vel_gpu(int nb_part, int tpb_part, int npart, float qom, float dx, float dt, float *Epx, float *u){
    update_vel<<<nb_part,tpb_part>>>(npart, qom, dx, dt, Epx, u);
}

void Init_gpu(Grid *grid, Fields *fields, Particles *part){
    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Jx, grid->get_ncell()*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Jy, grid->get_ncell()*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Jz, grid->get_ncell()*sizeof(float)) );

    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Ex, grid->get_ncell()*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Ey, grid->get_ncell()*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&fields->dev_Ez, grid->get_ncell()*sizeof(float)) );

    HANDLE_ERROR( hipMalloc( (void**)&part->dev_x, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_y, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_z, part->npart*sizeof(float)) );

    HANDLE_ERROR( hipMalloc( (void**)&part->dev_u, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_v, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_w, part->npart*sizeof(float)) );

    HANDLE_ERROR( hipMalloc( (void**)&part->dev_Ex, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_Ey, part->npart*sizeof(float)) );
    HANDLE_ERROR( hipMalloc( (void**)&part->dev_Ez, part->npart*sizeof(float)) );
}

void Stop_gpu(Fields *fields, Particles *part){
    HANDLE_ERROR( hipFree( fields->dev_Jx ) );
    HANDLE_ERROR( hipFree( fields->dev_Jy ) );
    HANDLE_ERROR( hipFree( fields->dev_Jz ) );

    HANDLE_ERROR( hipFree( fields->dev_Ex ) );
    HANDLE_ERROR( hipFree( fields->dev_Ey ) );
    HANDLE_ERROR( hipFree( fields->dev_Ez ) );

    HANDLE_ERROR( hipFree( part->dev_x ) );
    HANDLE_ERROR( hipFree( part->dev_y ) );
    HANDLE_ERROR( hipFree( part->dev_z ) );

    HANDLE_ERROR( hipFree( part->dev_u ) );
    HANDLE_ERROR( hipFree( part->dev_v ) );
    HANDLE_ERROR( hipFree( part->dev_w ) );

    HANDLE_ERROR( hipFree( part->dev_Ex ) );
    HANDLE_ERROR( hipFree( part->dev_Ey ) );
    HANDLE_ERROR( hipFree( part->dev_Ez ) );
}

void MemCpy_cpu2gpu(Grid *grid, Fields *fields, Particles *part){
    HANDLE_ERROR( hipMemcpy( fields->dev_Jx, fields->Jx, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( fields->dev_Jy, fields->Jy, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( fields->dev_Jz, fields->Jz, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemcpy( fields->dev_Ex, fields->Ex, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( fields->dev_Ey, fields->Ey, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( fields->dev_Ez, fields->Ez, grid->get_ncell()*sizeof(float), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemcpy( part->dev_x, part->x, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_y, part->y, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_z, part->z, part->npart*sizeof(float), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemcpy( part->dev_u, part->u, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_v, part->v, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_w, part->w, part->npart*sizeof(float), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemcpy( part->dev_Ex, part->Ex, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_Ey, part->Ey, part->npart*sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy( part->dev_Ez, part->Ez, part->npart*sizeof(float), hipMemcpyHostToDevice) );
}

void MemCpy_gpu2cpu(Grid *grid, Fields *fields, Particles *part){

    HANDLE_ERROR( hipMemcpy( fields->Jx, fields->dev_Jx, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( fields->Jy, fields->dev_Jy, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( fields->Jz, fields->dev_Jz, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy( fields->Ex, fields->dev_Ex, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( fields->Ey, fields->dev_Ey, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( fields->Ez, fields->dev_Ez, grid->get_ncell()*sizeof(float), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy( part->x, part->dev_x, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->y, part->dev_y, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->z, part->dev_z, part->npart*sizeof(float), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy( part->u, part->dev_u, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->v, part->dev_v, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->w, part->dev_w, part->npart*sizeof(float), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy( part->Ex, part->dev_Ex, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->Ey, part->dev_Ey, part->npart*sizeof(float), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy( part->Ez, part->dev_Ez, part->npart*sizeof(float), hipMemcpyDeviceToHost) );

}
